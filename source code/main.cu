#include "hip/hip_runtime.h"
#include "floating_number_helper.h"
#include "input_output.h"
#include <vector>
#include <time.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/copy.h>
#include <thrust/extrema.h>


//
// this function is what you need to finish
// @Usage : to solve the problem
// @Input : input containing all data needed
// @Output: answer containing all necessary data
//  you can find the definition of the two structs above in
//      input_output.h
//


//Second GPU version of 2D linear problem
//By Dong Yiming, Jul 26, 2018

#define MAXBOUND 1e20   //Initial left and right bound
#define MIN_NUMBER_OF_LINES 1000
#define EMPTYPOINT point(NAN,NAN)
#define EMPTYLINE line(0,0,0)

int begintime[100], endtime[100], duringtime[100];

//Only functions defined in .cu could be used, so I redefined them

__host__ __device__
inline int Strictly_less(double num1, double num2) {
	return (num1 + EPS < num2) ? TRUE : FALSE;
}

__host__ __device__
inline int Strictly_larger(double num1, double num2) {
	return (num1 - EPS > num2) ? TRUE : FALSE;
}

__host__ __device__
inline int Equals(double num1, double num2) {
	return fabs(num1 - num2) < EPS ? TRUE : FALSE;
}

__host__ __device__
double Compute_slope(line l) {
	if (Equals(l.param_b, 0)) {
		if (l.param_a > 0) {
			return -FLT_MAX;
		}
		return FLT_MAX;
	}
	return -l.param_a / l.param_b;
}

__host__ __device__
inline int Is_parallel(line line1, line line2) {
	return Equals(line1.param_a * line2.param_b, line1.param_b * line2.param_a);
}

__host__ __device__
point Generate_intersection_point(line line1, line line2) {
	point new_point;
	if (Is_parallel(line1, line2)) {
		return EMPTYPOINT;
	}
	else {
		new_point.pos_x = (line1.param_c * line2.param_b - line1.param_b * line2.param_c)
			/ (line1.param_a * line2.param_b - line1.param_b * line2.param_a);
		new_point.pos_y = (line1.param_c * line2.param_a - line1.param_a * line2.param_c)
			/ (line1.param_b * line2.param_a - line1.param_a * line2.param_b);
	}
	return new_point;
}

__host__ __device__
point Generate_intersection_point(line l, double boundary) {
	point new_point;
	if (Equals(l.param_b, 0)) {
		return EMPTYPOINT;
	}
	else {
		new_point.pos_x = boundary;
		new_point.pos_y = (l.param_c - l.param_a*boundary) / l.param_b;
	}
	return new_point;
}

struct compare_boundary_value {
	double boundary;
	compare_boundary_value(double _boundary) :boundary(_boundary) {}
	__host__ __device__
		bool operator()(line line1, line line2) {
		return Strictly_less(Generate_intersection_point(line1, boundary).pos_y, Generate_intersection_point(line2, boundary).pos_y);
	}
};

//Rotation and inverse-rotation functions
__host__ __device__
point rotation(point p, rotation_vector rv) {
	point rotatedpoint;
	rotatedpoint.pos_x = p.pos_x*rv.cosine - p.pos_y*rv.sine;
	rotatedpoint.pos_y = p.pos_x*rv.sine + p.pos_y*rv.cosine;
	return rotatedpoint;
}

__host__ __device__
line rotation(line l, rotation_vector rv) {
	line rotatedline;
	rotatedline.param_a = l.param_a*rv.cosine - l.param_b*rv.sine;
	rotatedline.param_b = l.param_a*rv.sine + l.param_b*rv.cosine;
	rotatedline.param_c = l.param_c;
	rotatedline.slope_value = Compute_slope(rotatedline);
	return rotatedline;
}

__host__ __device__
point inverse_rotation(point p, rotation_vector rv) {
	rv.sine = -rv.sine;
	return rotation(p, rv);
}

__host__ __device__
line inverse_rotation(line l, rotation_vector rv) {
	rv.sine = -rv.sine;
	return rotation(l, rv);
}

//Unary function of line rotation
struct rotate_lines {
	rotation_vector rv;
	rotate_lines(rotation_vector _rv) :rv(_rv) {}
	__host__ __device__
		line operator()(line paraline) {
		return rotation(paraline, rv);
	}
};

//Unary function of classfying lines into I+,I- or I0
struct in_Ipos {
	__host__ __device__
		bool operator()(line paraline) {
		return Strictly_larger(paraline.param_b, 0);
	}
};

struct in_Ineg {
	__host__ __device__
		bool operator()(line paraline) {
		return Strictly_less(paraline.param_b, 0);
	}
};

struct in_I0_left {
	__host__ __device__
		bool operator()(line paraline) {
		return Equals(paraline.param_b, 0) && Strictly_less(paraline.param_a, 0);
	}
};

struct in_I0_right {
	__host__ __device__
		bool operator()(line paraline) {
		return Equals(paraline.param_b, 0) && Strictly_larger(paraline.param_a, 0);
	}
};

struct line_to_boundary {
	__host__ __device__
		double operator()(line l) {
		return l.param_c / l.param_a;
	}
};

//Binary function of giving the criterion of comparing two lines to judge the new boundary
struct compare_line_boundary {
	__host__ __device__
		bool operator()(line line1, line line2) {
		return Strictly_less(line1.param_c / line1.param_a, line2.param_c / line2.param_a);
	}
};

//Determining whether the line is useful or redundant
struct useful_Ipos {
	point intersection_point;
	useful_Ipos(point _intersection_point) :intersection_point(_intersection_point) {}
	__host__ __device__
		bool operator()(line paraline) {
		return !Strictly_less(Generate_intersection_point(paraline, intersection_point.pos_x).pos_y, intersection_point.pos_y);
	}
};

struct useful_Ineg {
	point intersection_point;
	useful_Ineg(point _intersection_point) :intersection_point(_intersection_point) {}
	__host__ __device__
		bool operator()(line paraline) {
		return !Strictly_larger(Generate_intersection_point(paraline, intersection_point.pos_x).pos_y, intersection_point.pos_y);
	}
};



answer * compute(inputs * input) {

	int num = input->number;
	double objective_function_value;

	//All of the memory of object "ans" is allocated at this
	//And I don't use pointer to assign pointer, which may cause double free
	answer * ans = (answer *)malloc(sizeof(answer));
	ans->answer_b = DBL_MAX;
	ans->intersection_point = (point*)malloc(sizeof(point));
	ans->line1 = (line*)malloc(sizeof(line));
	ans->line2 = (line*)malloc(sizeof(line));

	//All lines transferred from *lines[] to vector

	thrust::host_vector <line>all_lines_host = input->lines;
	thrust::device_vector <line>all_lines_device = all_lines_host;

	//1.Rotate coordinary system
	//1.1 Construct the rotation vector

	rotation_vector rv;
	if (Equals(input->obj_function_param_a, 0)) {
		rv.cosine = 1; rv.sine = 0;
	}
	else if (Equals(input->obj_function_param_b, 0)) {
		rv.sine = 1; rv.cosine = 0;
	}
	else {
		rv.sine = input->obj_function_param_a / sqrt(input->obj_function_param_a*input->obj_function_param_a + input->obj_function_param_b*input->obj_function_param_b);
		rv.cosine = input->obj_function_param_b / sqrt(input->obj_function_param_a*input->obj_function_param_a + input->obj_function_param_b*input->obj_function_param_b);
	}

	//1.2 Rotate all the lines

	//thrust::device_vector <line>all_lines_device(num);
	thrust::transform(all_lines_device.begin(), all_lines_device.end(),
		all_lines_device.begin(), rotate_lines(rv));

	//1.3 Modify the objective function
	double rotated_obj_function_param_b = sqrt(input->obj_function_param_a*input->obj_function_param_a + input->obj_function_param_b*input->obj_function_param_b);
	//rotated_obj_function_param_a == 0


	//2.Classify lines into I+,I- and I0

	thrust::device_vector <line>rotated_lines_device_Ipos(num);
	thrust::device_vector <line>rotated_lines_device_Ineg(num);
	thrust::device_vector <line>rotated_lines_device_I0_left(num);
	thrust::device_vector <line>rotated_lines_device_I0_right(num);

	thrust::device_vector <line>::iterator Ipos_end = thrust::copy_if(all_lines_device.begin(), all_lines_device.end(), rotated_lines_device_Ipos.begin(), in_Ipos());
	thrust::device_vector <line>::iterator Ineg_end = thrust::copy_if(all_lines_device.begin(), all_lines_device.end(), rotated_lines_device_Ineg.begin(), in_Ineg());
	thrust::device_vector <line>::iterator I0_left_end = thrust::copy_if(all_lines_device.begin(), all_lines_device.end(), rotated_lines_device_I0_left.begin(), in_I0_left());
	thrust::device_vector <line>::iterator I0_right_end = thrust::copy_if(all_lines_device.begin(), all_lines_device.end(), rotated_lines_device_I0_right.begin(), in_I0_right());

	rotated_lines_device_Ipos.resize((Ipos_end - rotated_lines_device_Ipos.begin()));
	rotated_lines_device_Ineg.resize((Ineg_end - rotated_lines_device_Ineg.begin()));
	rotated_lines_device_I0_left.resize((I0_left_end - rotated_lines_device_I0_left.begin()));
	rotated_lines_device_I0_right.resize((I0_right_end - rotated_lines_device_I0_right.begin()));

	//3.Take the test line and remove lines
	//3.1 Initialize left and right boundary

	double left_boundary = -MAXBOUND, right_boundary = MAXBOUND;

	//Use I0_left to update left boundary
	if (!rotated_lines_device_I0_left.empty()) {
		thrust::device_vector <line>::iterator max_left_boundary_iter = thrust::max_element(rotated_lines_device_I0_left.begin(), rotated_lines_device_I0_left.end(), compare_line_boundary());
		thrust::host_vector<line> max_left_boundary_vector(max_left_boundary_iter, max_left_boundary_iter + 1);
		left_boundary = max_left_boundary_vector[0].param_c / max_left_boundary_vector[0].param_a;
	}

	//Use I0_right to update right boundary
	if (!rotated_lines_device_I0_right.empty()) {
		thrust::device_vector <line>::iterator min_right_boundary_iter = thrust::min_element(rotated_lines_device_I0_right.begin(), rotated_lines_device_I0_right.end(), compare_line_boundary());
		thrust::host_vector<line> min_right_boundary_vector(min_right_boundary_iter, min_right_boundary_iter + 1);
		right_boundary = min_right_boundary_vector[0].param_c / min_right_boundary_vector[0].param_a;
	}

	//Remove redundant lines: keep removing if the number of lines is not smaller than a limiteded size

	while (rotated_lines_device_Ipos.size() + rotated_lines_device_Ineg.size() >= MIN_NUMBER_OF_LINES) {
		//3.2 Determine the line in I+ and I- with largest/smallest y-axis of intersection point with boundaries respectively


		//Calculate the max boundary lines in I+ and I- and get the intersection point
		line max_left_boundary_line = *thrust::max_element(rotated_lines_device_Ipos.begin(), rotated_lines_device_Ipos.end(), compare_boundary_value(left_boundary));
		line max_right_boundary_line = *thrust::max_element(rotated_lines_device_Ipos.begin(), rotated_lines_device_Ipos.end(), compare_boundary_value(right_boundary));
		point intersection_point_Ipos = Generate_intersection_point(max_left_boundary_line, max_right_boundary_line);


		line min_left_boundary_line = *thrust::min_element(rotated_lines_device_Ineg.begin(), rotated_lines_device_Ineg.end(), compare_boundary_value(left_boundary));
		line min_right_boundary_line = *thrust::min_element(rotated_lines_device_Ineg.begin(), rotated_lines_device_Ineg.end(), compare_boundary_value(right_boundary));
		point intersection_point_Ineg = Generate_intersection_point(min_left_boundary_line, min_right_boundary_line);

		//3.3 Generate test line
		double test_line_Ipos, test_line_Ineg;
		if (intersection_point_Ipos == EMPTYPOINT)
			test_line_Ipos = NAN;
		else test_line_Ipos = intersection_point_Ipos.pos_x;
		if (intersection_point_Ineg == EMPTYPOINT)
			test_line_Ineg = NAN;
		else test_line_Ineg = intersection_point_Ineg.pos_x;

		//3.4 Remove redundant lines

		thrust::device_vector <line>::iterator Ipos_useful_end = thrust::copy_if(rotated_lines_device_Ipos.begin(), rotated_lines_device_Ipos.end(), rotated_lines_device_Ipos.begin(), useful_Ipos(intersection_point_Ipos));
		thrust::device_vector <line>::iterator Ineg_useful_end = thrust::copy_if(rotated_lines_device_Ineg.begin(), rotated_lines_device_Ineg.end(), rotated_lines_device_Ineg.begin(), useful_Ineg(intersection_point_Ineg));
		rotated_lines_device_Ipos.resize(Ipos_useful_end - rotated_lines_device_Ipos.begin());
		rotated_lines_device_Ineg.resize(Ineg_useful_end - rotated_lines_device_Ineg.begin());

		//3.5 Mark the test line as the new boundary
		line max_Ipos_line_of_Ipos_testline = *thrust::max_element(rotated_lines_device_Ipos.begin(), rotated_lines_device_Ipos.end(), compare_boundary_value(test_line_Ipos));
		line min_Ineg_line_of_Ipos_testline = *thrust::min_element(rotated_lines_device_Ineg.begin(), rotated_lines_device_Ineg.end(), compare_boundary_value(test_line_Ipos));
		line max_Ipos_line_of_Ineg_testline = *thrust::max_element(rotated_lines_device_Ipos.begin(), rotated_lines_device_Ipos.end(), compare_boundary_value(test_line_Ineg));
		line min_Ineg_line_of_Ineg_testline = *thrust::min_element(rotated_lines_device_Ineg.begin(), rotated_lines_device_Ineg.end(), compare_boundary_value(test_line_Ineg));

		//The direction of optimal solution can be decided as follows by the slope of line in I+:
		//if slope<0 then the optimal solution in on the right, thus the test line maybe the left boundary
		//if slope>0 then the optimal solution in on the left, thus the test line maybe the right boundary

		//After knowing the direction, compare the test line and current boundary to decide whether we modify the boundary
		if (Strictly_less(Generate_intersection_point(max_Ipos_line_of_Ipos_testline, test_line_Ipos).pos_y, Generate_intersection_point(min_Ineg_line_of_Ipos_testline, test_line_Ipos).pos_y)) {
			if (Strictly_larger(max_Ipos_line_of_Ipos_testline.slope_value, 0) && Strictly_less(test_line_Ipos, right_boundary)) {
				right_boundary = test_line_Ipos;
			}
			else if (Strictly_less(max_Ipos_line_of_Ipos_testline.slope_value, 0) && Strictly_larger(test_line_Ipos, left_boundary)) {
				left_boundary = test_line_Ipos;
			}
		}
		else {
			if (Strictly_larger(max_Ipos_line_of_Ipos_testline.slope_value, min_Ineg_line_of_Ipos_testline.slope_value) && Strictly_less(test_line_Ipos, right_boundary))
				right_boundary = test_line_Ipos;
			else if (Strictly_less(max_Ipos_line_of_Ipos_testline.slope_value, min_Ineg_line_of_Ipos_testline.slope_value) && Strictly_larger(test_line_Ipos, left_boundary))
				left_boundary = test_line_Ipos;
		}


		if (Strictly_less(Generate_intersection_point(max_Ipos_line_of_Ineg_testline, test_line_Ineg).pos_y, Generate_intersection_point(min_Ineg_line_of_Ineg_testline, test_line_Ineg).pos_y)) {
			if (Strictly_larger(max_Ipos_line_of_Ineg_testline.slope_value, 0) && Strictly_less(test_line_Ineg, right_boundary)) {
				right_boundary = test_line_Ineg;
			}
			else if (Strictly_less(max_Ipos_line_of_Ineg_testline.slope_value, 0) && Strictly_larger(test_line_Ineg, left_boundary)) {
				left_boundary = test_line_Ineg;
			}
		}
		else {
			if (Strictly_larger(max_Ipos_line_of_Ineg_testline.slope_value, min_Ineg_line_of_Ineg_testline.slope_value) && Strictly_less(test_line_Ineg, right_boundary))
				right_boundary = test_line_Ineg;
			else if (Strictly_less(max_Ipos_line_of_Ineg_testline.slope_value, min_Ineg_line_of_Ineg_testline.slope_value) && Strictly_larger(test_line_Ineg, left_boundary))
				left_boundary = test_line_Ineg;
		}
	}

	//After the circulation, now get the useful lines
	std::vector<line> useful_lines;
	useful_lines.insert(useful_lines.end(), rotated_lines_device_Ipos.begin(), rotated_lines_device_Ipos.end());
	useful_lines.insert(useful_lines.end(), rotated_lines_device_Ineg.begin(), rotated_lines_device_Ineg.end());

	//4.Use CPU to solve the small problem which involves less than 10 lines
	for (int i = 0; i < useful_lines.size(); i++)
		for (int j = i + 1; j < useful_lines.size(); j++) {
			bool flag = TRUE;
			point new_point = Generate_intersection_point(useful_lines[i], useful_lines[j]);
			if (!(new_point == EMPTYPOINT)) {
				for (int k = 0; k < useful_lines.size(); k++) {
					if (Strictly_less(useful_lines[k].param_a*new_point.pos_x + useful_lines[k].param_b*new_point.pos_y, useful_lines[k].param_c))
						flag = FALSE;
				}
				objective_function_value = new_point.pos_y*sqrt(input->obj_function_param_a*input->obj_function_param_a + input->obj_function_param_b*input->obj_function_param_b);
				if (strictly_less(objective_function_value, ans->answer_b) && flag) {
					//5.Rotate back the lines and point if the objective function value of new_point is less than current value
					ans->answer_b = objective_function_value;
					*ans->line1 = inverse_rotation(line(useful_lines[i].param_a, useful_lines[i].param_b, useful_lines[i].param_c, useful_lines[i].slope_value), rv);
					*ans->line2 = inverse_rotation(line(useful_lines[j].param_a, useful_lines[j].param_b, useful_lines[j].param_c, useful_lines[j].slope_value), rv);
					*ans->intersection_point = inverse_rotation(new_point, rv);
				}
			}
		}
	return ans;
}


int main() {
	int inputfilename[] = { 15000000};
	FILE* output_file = fopen("../result.dat", "w");
	for (auto i : inputfilename) {
		// 1. get the input data
		//inputs * input = read_from_file("../1000000_0.dat");
		char a[50] = "../"; char b[50]; itoa(i, b, 10); char* c = "_0.dat"; strcat(b, c); strcat(a, b);
		inputs * input = read_from_file(a);
		// 2. get the answer
		answer * ans = compute(input);
		for (int i = 0; i < 100; i++) {
			begintime[i] = clock();
			ans = compute(input);
			endtime[i] = clock();
		}
		// 3. display result and free memory
		char * ans_string = generate_ans_string(ans);
		printf("%s", ans_string);
		for (int i = 0; i < 100; i++) {
			duringtime[i] = endtime[i] - begintime[i];
			fprintf(output_file, "%d\t", duringtime[i]);
			printf("time:%d\t", duringtime[i]);
		}
		printf("\n\n");
	
		free_inputs(&input);
		free_ans(&ans);
		free(ans_string);
	}	
	fclose(output_file);
	return 0;
}
